#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>


const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)


// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
   for (int i = 0; i<DSIZE; i++){
      for(int j=0; j<DSIZE;j++){
         for(int k;k<DSIZE;k++){
         C[i][j] += A[i][k] * B[k][j];
        }
      }
    } 
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {
  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            sum += A[idy*size + i] * B[i*size + idx];
        }
        C[idy*size+idx] = temp;                    
    }

}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
   
    float cpu_A[DSIZE][DSIZE];
    float cpu_B[DSIZE][DSIZE];
   
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }
   
    //Initialize 2d matrix cpu_A
    for (int i = 0; i<DSIZE; i++){
      for(int j=0; j<DSIZE;j++){
         cpu_A[i][j] = A_val;
      }
    } 
   
   //Initialize 2d matrix cpu_B
   for (int i = 0; i<DSIZE; i++){
      for(int j=0; j<DSIZE;j++){
         cpu_B[i][j] = B_val;
      }
    } 
   
   //Initialize 2d matric cpu_C
   for (int i = 0; i<DSIZE; i++){
      for(int j=0; j<DSIZE;j++){
         cpu_C[i][j] = 0;
      }
    } 
   
    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
   
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE*sizeof(float), hipMemcpyHostToDevice);
   
    // Launch kernel
    // Specify the block and grid dimentions 
    dim3 block(32,32);
    dim3 grid(8,8); 
    matrix_mul_gpu<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

    cudaCheckErrors("O no.");
   
    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    matrix_mul_cpu(cpu_A, cpu_B, cpu_C, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t3sum);

    printf ("Checking multiplication GPU (1d array): \n");
    for (int i = 0; i<5; i++){
       printf("This is element h_C[%d]: %f\n",i,h_C[i]);
     } 
   
    printf ("\nChecking multiplication CPU (2d matrix): \n");
    for (int i = 0; i<5; i++){
       for(int j=0; j<5;j++){
          printf("This is element cpu_[%d][%d]: %f\n",i,j,cpu_C[i][j]);
       }
     } 

    // Free memory 
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;

}
