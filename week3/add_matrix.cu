#include "hip/hip_runtime.h"
#include <stdio.h>


const int DSIZE_X = 256;
const int DSIZE_Y = 256;

__global__ void add_matrix(float A[DSIZE_X][DSIZE_Y], float B[DSIZE_X][DSIZE_Y], float C[DSIZE_X][DSIZE_Y])
{
    //FIXME:
    // Express in terms of threads and blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    // Add the two matrices - make sure you are not out of range
    if (idx < DSIZE_X && idy < DSIZE_Y){
        
        C[idx][idy] = A[idx][idy] + B[idx][idy];
    }

}

int main()
{

    // Create and allocate memory for host and device pointers 
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, DSIZE_X * DSIZE_Y * sizeof(float));
    hipMalloc((void**)&d_B, DSIZE_X * DSIZE_Y * sizeof(float));
    hipMalloc((void**)&d_C, DSIZE_X * DSIZE_Y * sizeof(float));
    h_A = new float[DSIZE_X * DSIZE_Y];
    h_B = new float[DSIZE_X * DSIZE_Y];
    h_C = new float[DSIZE_X * DSIZE_Y];

    // Fill in the matrices
    for (int i = 0; i < DSIZE_X * DSIZE_Y; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }
    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    // dim3 is a built in CUDA type that allows you to define the block 
    // size and grid size in more than 1 dimentions
    // Syntax : dim3(Nx,Ny,Nz)
    dim3 blockSize(16, 16);
    dim3 gridSize((DSIZE_X + blockSize.x - 1) / blockSize.x, (DSIZE_Y + blockSize.y - 1) / blockSize.y);

    // Copy back to host
    hipMemcpy(h_C, d_C, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make the addition was succesfull
    for (int i = 0; i < 10; i++) {
        int x = rand() % DSIZE_X;
        int y = rand() % DSIZE_Y;
        printf("A[%d][%d] + B[%d][%d] = C[%d][%d] -> %f + %f = %f\n", x, y, x, y, x, y, h_A[x * DSIZE_Y + y], h_B[x * DSIZE_Y + y], h_C[x * DSIZE_Y + y]);
    }
    // Free the memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}