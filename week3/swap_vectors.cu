
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_addition(float *array_A, float *array_B) {

    //FIXME:
    // Express the vector index in terms of threads and blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Swap the vector elements - make sure you are not out of range
    if (idx < DSIZE) {
        float temp = array_A[idx];
        array_A[idx] = array_B[idx];
        array_B[idx] = temp;
    }

}


int main() {


    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }


    // Allocate memory for host and device pointers 
    hipMalloc((void**)&d_A, DSIZE * sizeof(float));
    hipMalloc((void**)&d_B, DSIZE * sizeof(float));
    hipMalloc((void**)&d_C, DSIZE * sizeof(float));
    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    // Launch the kernel
    vector_addition<<<grid_size, block_size>>>(d_A, d_B);
    // Copy back to host
    hipMemcpy(h_A, d_A, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    // Print and check some elements to make sure swapping was successfull
    for (int i = 0; i < 10; i++) {
        printf("A[%d] = %f, B[%d] = %f\n", i, h_A[i], i, h_B[i]);
    }
    // Free the memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
